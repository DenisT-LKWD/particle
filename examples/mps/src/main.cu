#include "hip/hip_runtime.h"
// Particle headers
#include <particle/collection.hpp>
#include <particle/geometry/adapted/cuda/float2.hpp>
#include <particle/geometry/adapted/cuda/int2.hpp>
#include <particle/geometry/adapted/cuda/thrust_tuple.hpp>
#include <particle/geometry/adapted/cuda/thrust_zip_iterator.hpp>
#include <particle/geometry/box.hpp>
#include <particle/grid/left_grid.hpp>
#include <particle/io/vtk.hpp>
#include <particle/neighbour/cell_list.hpp>

// Thrust headers
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/host_vector.h>

// Boost headers
#include <boost/chrono.hpp>
#include <boost/mpl/vector.hpp>
#include <boost/program_options.hpp>
#include <boost/property_tree/ptree.hpp>
#include <boost/property_tree/xml_parser.hpp>
#include <boost/random.hpp>

// Std headers
#include <iostream>
#include <vector>

struct input_params
{
  std::size_t particle_count;
  float cutoff;
  float lower_x;
  float upper_x;
  float lower_y;
  float upper_y;
  float lower_z;
  float upper_z;
  float cell_size;
  std::size_t step_count;
  
  void load(const std::string &filename)
  {
    namespace pt = boost::property_tree;

    pt::ptree tree;
    pt::read_xml(filename, tree);
    particle_count = tree.get<std::size_t>("input.count");
    cutoff = tree.get<float>("input.cutoff");
    lower_x = tree.get<float>("input.lower.x");
    lower_y = tree.get<float>("input.lower.y");
    lower_z = tree.get<float>("input.lower.z");
    upper_x = tree.get<float>("input.upper.x");
    upper_y = tree.get<float>("input.upper.y");
    upper_z = tree.get<float>("input.upper.z");
    cell_size = tree.get<float>("input.cell_size");
    step_count = tree.get<std::size_t>("input.step_count");
  }

  void save(const std::string &filename)
  {
    namespace pt = boost::property_tree;

    pt::ptree tree;
    tree.put("input.count", particle_count);
    tree.put("input.cutoff", cutoff);
    tree.put("input.lower.x", lower_x);
    tree.put("input.lower.y", lower_y);
    tree.put("input.lower.z", lower_z);
    tree.put("input.upper.x", upper_x);
    tree.put("input.upper.y", upper_y);
    tree.put("input.upper.z", upper_z);
    tree.put("input.cell_size", cell_size);
    tree.put("input.step_count", step_count);
    pt::write_xml(filename, tree);
  }
};

template <class ForwardIterator>
void random_fill(ForwardIterator begin, ForwardIterator end, double lower, double upper)
{
  static boost::random::mt11213b rng;
  boost::random::uniform_real_distribution<> dist(lower, upper);
  std::generate(begin, end, [&]() { return dist(rng); });
}

struct id_;
struct type_;
struct x_;
struct y_;
struct vx_;
struct vy_;

using position_ = boost::mpl::vector<x_, y_>;
using velocity_ = boost::mpl::vector<vx_, vy_>;

int main(int argc, char** argv)
{
  namespace po = boost::program_options;
  
  std::string filename;
  po::options_description desc("Allowed options");
  desc.add_options()
    ("help", "produce help message")
    ("touch", "produce empty input file")
    ("input,i", po::value<std::string>(&filename), "set input filename")
    ;

  po::variables_map vm;
  po::store(po::parse_command_line(argc, argv, desc), vm);
  po::notify(vm);

  if (vm.count("touch"))
  {
    input_params ip;
    ip.save("input.xml");
    return 1;
  }

  if (vm.count("help") || filename.empty()) {
    std::cout << desc << std::endl;
    return 1;
  }
  
  input_params ip;
  ip.load(filename);
  
  using boost::fusion::pair;
  particle::collection<
    pair<id_, thrust::device_vector<int>>
    , pair<type_, thrust::device_vector<int>>
    , pair<x_, thrust::device_vector<float>>
    , pair<y_, thrust::device_vector<float>>> col;
  col.resize(ip.particle_count);

  thrust::host_vector<float> x(ip.particle_count);
  random_fill(x.begin(), x.end(), ip.lower_x, ip.upper_x);

  thrust::host_vector<float> y(ip.particle_count);
  random_fill(y.begin(), y.end(), ip.lower_y, ip.upper_y);

  thrust::host_vector<int> id(ip.particle_count);
  thrust::host_vector<int> type(ip.particle_count);
  for (int i = 0; i < ip.particle_count; i++)
  {
    id[i] = i;
    type[i] = 2 * i / ip.particle_count;
  }

  thrust::copy(id.begin(), id.end(), col.begin<id_>());
  thrust::copy(type.begin(), type.end(), col.begin<type_>());
  thrust::copy(x.begin(), x.end(), col.begin<x_>());
  thrust::copy(y.begin(), y.end(), col.begin<y_>());
  
  thrust::for_each(col.begin(), col.end(),
                   PARTICLE_LAMBDA (const thrust::tuple<int, int, float, float>& x)
                   { printf("%d %d %f %f\n", thrust::get<0>(x), thrust::get<1>(x), thrust::get<2>(x), thrust::get<3>(x)); });

  using box_type = particle::geometry::box<int2, int2>;

  auto grid = particle::left_grid<box_type, float>(
    box_type(
      make_int2(std::floor(ip.lower_x / ip.cell_size) - 1,
                std::floor(ip.lower_y / ip.cell_size) - 1)
      , make_int2(std::ceil(ip.upper_x / ip.cell_size) + 1,
                  std::ceil(ip.upper_y / ip.cell_size) + 1))
    , ip.cell_size);

  particle::neighbour::cell_list<decltype(grid)> cl(grid);
  cl.build(col.begin<position_>(), col.end<position_>(), col.begin());

  thrust::for_each(col.begin(), col.end(),
                   PARTICLE_LAMBDA (const thrust::tuple<int, int, float, float>& x)
                   { printf("%d %d %f %f\n", thrust::get<0>(x), thrust::get<1>(x), thrust::get<2>(x), thrust::get<3>(x)); });

  cl.pair_interact(
    col.begin<position_>()
    , col.end<position_>()
    , PARTICLE_LAMBDA(int i, int j)
    { if (i != j) printf("%d %d\n", i, j); });
  
  particle::collection<
    pair<id_, thrust::host_vector<int>>
    , pair<type_, thrust::host_vector<int>>
    , pair<x_, thrust::host_vector<float>>
    , pair<y_, thrust::host_vector<float>>> col_host;
  col_host.resize(ip.particle_count);
  thrust::copy(col.begin(), col.end(), col_host.begin());

  particle::io::vtk::save(
    "test.vtu"
    , col_host.begin<position_>()
    , col_host.end<position_>()
    , boost::fusion::make_vector("id", "type")
    , boost::fusion::make_vector(col_host.begin<id_>(), col_host.begin<type_>())); 

  return 0;
}
