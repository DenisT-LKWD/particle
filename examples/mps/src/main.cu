#include "hip/hip_runtime.h"
// Particle headers
#include <particle/collection.hpp>
#include <particle/geometry/adapted/cuda/double2.hpp>
#include <particle/geometry/adapted/cuda/int2.hpp>
#include <particle/geometry/adapted/cuda/thrust_tuple.hpp>
#include <particle/geometry/adapted/cuda/thrust_zip_iterator.hpp>
#include <particle/geometry/as.hpp>
#include <particle/geometry/box.hpp>
#include <particle/geometry/distance.hpp>
#include <particle/geometry/elem.hpp>
#include <particle/geometry/length.hpp>
#include <particle/geometry/operators.hpp>
#include <particle/grid/left_grid.hpp>
#include <particle/io/vtk.hpp>
#include <particle/mps/kernels.hpp>
#include <particle/mps/lambda.hpp>
#include <particle/mps/pnd0.hpp>
#include <particle/neighbour/cell_list.hpp>

// Thrust headers
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/host_vector.h>

// Boost headers
#include <boost/chrono.hpp>
#include <boost/foreach.hpp>
#include <boost/mpl/vector.hpp>
#include <boost/program_options.hpp>
#include <boost/property_tree/ptree.hpp>
#include <boost/property_tree/xml_parser.hpp>
#include <boost/random.hpp>

// Std headers
#include <iostream>
#include <vector>

enum class category_t {fluid, solid, ghost, unknown};
enum class motion_t {free, fixed, imposed, unknown};

struct input_params
{
  struct type
  {
    std::string name;
    category_t category;
    motion_t motion;
    float density;
    float viscosity;
  };

  struct box
  {
    std::string type;
    float origin[3];
    int size[3];

    box(): origin{0, 0, 0}, size{0, 0, 0} {}
  };

  float time_step;
  float distance_between_particles;
  float cutoff;
  float gravity;
  float speed_of_sound;
  float free_surface_threshold;
  float free_surface_pressure;
  float lower[3];
  float upper[3];
  float cell_size;
  std::size_t step_count;
  std::size_t steps_per_output;
  std::map<std::string, type> types;
  std::vector<box> boxes;
  std::string output_filename;
  
  input_params()
    : time_step(1)
    , distance_between_particles(0)
    , cutoff(0)
    , gravity(0)
    , speed_of_sound(0)
    , free_surface_threshold(0)
    , free_surface_pressure(0)
    , lower{0, 0, 0}
    , upper{0, 0, 0}
    , cell_size(0)
    , step_count(0)
    , steps_per_output(1)
  {}
  
  void load(const std::string &filename)
  {
    namespace pt = boost::property_tree;

    pt::ptree tree;
#pragma push
#pragma diag_suppress = code_is_unreachable
#pragma diag_suppress = initialization_not_reachable
    pt::read_xml(filename, tree, pt::xml_parser::trim_whitespace);
#pragma pop
    time_step = tree.get<float>("input.time_step");
    distance_between_particles = tree.get<float>("input.distance_between_particles");
    cutoff = tree.get<float>("input.cutoff");
    gravity = tree.get<float>("input.gravity");
    speed_of_sound = tree.get<float>("input.speed_of_sound");
    free_surface_threshold = tree.get<float>("input.free_surface_threshold");
    free_surface_pressure = tree.get<float>("input.free_surface_pressure");
    lower[0] = tree.get<float>("input.lower.x");
    lower[1] = tree.get<float>("input.lower.y");
    lower[2]= tree.get<float>("input.lower.z");
    upper[0] = tree.get<float>("input.upper.x");
    upper[1]= tree.get<float>("input.upper.y");
    upper[2] = tree.get<float>("input.upper.z");
    cell_size = tree.get<float>("input.cell_size");
    step_count = tree.get<std::size_t>("input.step_count");
    steps_per_output = tree.get<std::size_t>("input.steps_per_output");
    BOOST_FOREACH (auto& c, tree.get_child("input").equal_range("type"))
    {
      type t;
      t.name = c.second.get<std::string>("name");
      std::string category = c.second.get<std::string>("category");
      if (category == "fluid")
        t.category = category_t::fluid;
      else if (category == "solid")
        t.category = category_t::solid;
      else if (category == "ghost")
        t.category = category_t::ghost;
      else
        t.category = category_t::unknown;
      std::string motion = c.second.get<std::string>("motion");
      if (motion == "free")
        t.motion = motion_t::free;
      else if (motion == "fixed")
        t.motion = motion_t::fixed;
      else if (motion == "imposed")
        t.motion = motion_t::imposed;
      else
        t.motion = motion_t::unknown;
      t.density = c.second.get<float>("density");
      t.viscosity = c.second.get<float>("viscosity");
      types.insert({t.name, t});
    }
    try
    {
      BOOST_FOREACH (auto& c, tree.get_child("input").equal_range("box"))
      {
        box b;
        b.type = c.second.get<std::string>("type");
        b.origin[0] = c.second.get<float>("origin_x");
        b.origin[1] = c.second.get<float>("origin_y");
        b.origin[2] = c.second.get<float>("origin_z");
        b.size[0] = c.second.get<int>("size_x");
        b.size[1] = c.second.get<int>("size_y");
        b.size[2] = c.second.get<int>("size_z");
        boxes.push_back(b);
      }
    }
    catch(...) {} // no boxes on input file
    output_filename = tree.get<std::string>("input.output_filename");
  }
};

template <class ForwardIterator>
void random_fill(ForwardIterator begin, ForwardIterator end, double lower, double upper)
{
  static boost::random::mt11213b rng;
  boost::random::uniform_real_distribution<> dist(lower, upper);
  std::generate(begin, end, [&]() { return dist(rng); });
}

namespace tag
{
  struct id;
  struct type;
  struct x;
  struct y;
  struct vx;
  struct vy;
  struct ax;
  struct ay;
  struct pnd;
  struct pressure;
  struct pressure_grad_x;
  struct pressure_grad_y;
  struct pred_x;
  struct pred_y;
  struct pred_vx;
  struct pred_vy;
  
  using position = boost::mpl::vector<x, y>;
  using velocity = boost::mpl::vector<vx, vy>;
  using acceleration = boost::mpl::vector<ax, ay>;
  using pressure_grad = boost::mpl::vector<pressure_grad_x, pressure_grad_y>;
  using pred_position = boost::mpl::vector<pred_x, pred_y>;
  using pred_velocity = boost::mpl::vector<pred_vx, pred_vy>;
}

const int max_types = 10;
__constant__ int categories[max_types];
__constant__ int motions[max_types];
__constant__ double densities[max_types];
__constant__ double viscosities[max_types];

int main(int argc, char** argv)
{
  namespace po = boost::program_options;
  using particle::geometry::operators::operator+;
  using particle::geometry::operators::operator-;
  using particle::geometry::operators::operator*;
  using particle::geometry::operators::operator/;
  
  std::string filename;
  po::options_description desc("Allowed options");
  desc.add_options()
    ("help", "produce help message")
    ("input,i", po::value<std::string>(&filename), "set input filename")
    ;

  po::variables_map vm;
  po::store(po::parse_command_line(argc, argv, desc), vm);
  po::notify(vm);

  if (vm.count("help") || filename.empty()) {
    std::cout << desc << std::endl;
    return 0;
  }
  
  input_params ip;
  ip.load(filename);

  // Generate boxes
  thrust::host_vector<int> id;
  thrust::host_vector<int> type;
  thrust::host_vector<double> x;
  thrust::host_vector<double> y;
  int particle_count = 0;
  for (auto &b : ip.boxes)
  {
    int t = std::distance(ip.types.begin(), ip.types.find(b.type));
    double2 origin = make_double2(b.origin[0], b.origin[1]);
    for (int j = 0; j < b.size[1]; j++)
    {
      for (int i = 0; i < b.size[0]; i++)
      {
        id.push_back(particle_count++);
        type.push_back(t);
        auto p = origin + ip.distance_between_particles * make_double2(i, j);
        x.push_back(particle::geometry::elem<0>(p));
        y.push_back(particle::geometry::elem<1>(p));
      }
    }
  }

  using boost::fusion::pair;
  particle::collection<
    pair<tag::id, thrust::device_vector<int>>
    , pair<tag::type, thrust::device_vector<int>>
    , pair<tag::x, thrust::device_vector<double>>
    , pair<tag::y, thrust::device_vector<double>>
    , pair<tag::vx, thrust::device_vector<double>>
    , pair<tag::vy, thrust::device_vector<double>>
    , pair<tag::ax, thrust::device_vector<double>>
    , pair<tag::ay, thrust::device_vector<double>>
    , pair<tag::pnd, thrust::device_vector<double>>
    , pair<tag::pressure, thrust::device_vector<double>>
    , pair<tag::pressure_grad_x, thrust::device_vector<double>>
    , pair<tag::pressure_grad_y, thrust::device_vector<double>>
    , pair<tag::pred_x, thrust::device_vector<double>>
    , pair<tag::pred_y, thrust::device_vector<double>>
    , pair<tag::pred_vx, thrust::device_vector<double>>
    , pair<tag::pred_vy, thrust::device_vector<double>>
    > col;
  col.resize(particle_count);

  particle::collection<
    pair<tag::id, thrust::host_vector<int>>
    , pair<tag::type, thrust::host_vector<int>>
    , pair<tag::x, thrust::host_vector<double>>
    , pair<tag::y, thrust::host_vector<double>>
    , pair<tag::vx, thrust::host_vector<double>>
    , pair<tag::vy, thrust::host_vector<double>>
    , pair<tag::pnd, thrust::host_vector<double>>
    , pair<tag::pressure, thrust::host_vector<double>>
    > h_col;
  h_col.resize(particle_count);
  
  thrust::copy(id.begin(), id.end(), col.begin<tag::id>());
  thrust::copy(type.begin(), type.end(), col.begin<tag::type>());
  thrust::copy(x.begin(), x.end(), col.begin<tag::x>());
  thrust::copy(y.begin(), y.end(), col.begin<tag::y>());
  thrust::fill(
    col.begin<tag::velocity>()
    , col.end<tag::velocity>()
    , thrust::make_tuple(0.0, 0.0));
  thrust::fill(col.begin<tag::pnd>(), col.end<tag::pnd>(), 0);
  thrust::fill(col.begin<tag::pressure>(), col.end<tag::pressure>(), 0);
  
  using box_type = particle::geometry::box<int2, int2>;
  auto grid = particle::left_grid<box_type, double>(
    box_type(
      make_int2(std::floor(ip.lower[0] / ip.cell_size) - 1,
                std::floor(ip.lower[1] / ip.cell_size) - 1)
      , make_int2(std::ceil(ip.upper[0] / ip.cell_size) + 1,
                  std::ceil(ip.upper[1] / ip.cell_size) + 1))
    , ip.cell_size);

  using cell_list = particle::neighbour::cell_list<decltype(grid)>;
  cell_list cl(grid);

  const int dimensions = 2;
  const auto kernel = particle::mps::rational_kernel<double>(ip.cutoff);
  const double dbp = ip.distance_between_particles;
  const double pnd0 = particle::mps::pnd0<2>(dbp, static_cast<double>(ip.cutoff), kernel);
  const double lambda = particle::mps::lambda<2>(dbp, static_cast<double>(ip.cutoff), pnd0, kernel);

  const double laplacian_constant = 2 * dimensions / (lambda * pnd0);
  const double gradient_constant = dimensions / pnd0;

  int h_categories[max_types];
  std::transform(
    ip.types.begin()
    , ip.types.end()
    , h_categories
    , [](auto t) { return static_cast<int>(t.second.category); });
  hipMemcpyToSymbol(HIP_SYMBOL(categories), h_categories, ip.types.size() * sizeof(int));

  int h_motions[max_types];
  std::transform(
    ip.types.begin()
    , ip.types.end()
    , h_motions
    , [](auto t) { return static_cast<int>(t.second.motion); });
  hipMemcpyToSymbol(HIP_SYMBOL(motions), h_motions, ip.types.size() * sizeof(int));

  double h_densities[max_types];
  std::transform(
    ip.types.begin()
    , ip.types.end()
    , h_densities
    , [](auto t) { return t.second.density; });
  hipMemcpyToSymbol(HIP_SYMBOL(densities), h_densities, ip.types.size() * sizeof(double));

  double h_viscosities[max_types];
  std::transform(
    ip.types.begin()
    , ip.types.end()
    , h_viscosities
    , [](auto t) { return t.second.viscosity; });
  hipMemcpyToSymbol(HIP_SYMBOL(viscosities), h_viscosities, ip.types.size() * sizeof(double));

  using output_tags = boost::mpl::vector<tag::id, tag::type, tag::x, tag::y, tag::vx, tag::vy, tag::pnd, tag::pressure>;

  {
    thrust::copy(col.begin<output_tags>(), col.end<output_tags>(), h_col.begin<output_tags>());    std::stringstream ss;
    ss << ip.output_filename << "_0.vtu";
    particle::io::vtk::save(
      ss.str()
      , h_col.begin<tag::position>()
      , h_col.end<tag::position>()
      , boost::fusion::make_vector("id", "type", "pnd", "pressure", "velocity")
      , boost::fusion::make_vector(
        h_col.begin<tag::id>()
        , h_col.begin<tag::type>()
        , h_col.begin<tag::pnd>()
        , h_col.begin<tag::pressure>()
        , h_col.begin<tag::velocity>()));
  }

  // Simulation loop
  for (int step = 0; step < ip.step_count; step++)
  {
    cl.build(
      col.begin<tag::position>()
      , col.end<tag::position>()
      , col.begin<boost::mpl::vector<
      tag::id, tag::type, tag::x, tag::y, tag::vx, tag::vy>>());

    double gravity = ip.gravity;
    thrust::transform(
      col.begin<tag::type>()
      , col.end<tag::type>()
      , col.begin<tag::acceleration>()
      , PARTICLE_LAMBDA_DEVICE(int t)
      {
        if (motions[t] == static_cast<int>(motion_t::free))
          return thrust::make_tuple(0.0, -gravity);
        return thrust::make_tuple(0.0, 0.0);
      });

    auto type_first = col.begin<tag::type>();
    auto position_first = col.begin<tag::position>();
    auto velocity_first = col.begin<tag::velocity>();
    auto acceleration_first = col.begin<tag::acceleration>();
    cl.pair_interact(
      col.begin<tag::position>()
      , col.end<tag::position>()
      , PARTICLE_LAMBDA_DEVICE(int i, cell_list::neighbour_list nl)
      {
        using particle::geometry::as;
        int type_i = *(type_first + i);
        if (categories[type_i] != static_cast<int>(category_t::fluid))
          return;
        auto pos_i = *(position_first + i);
        auto vel_i = *(velocity_first + i);
        auto acc_i = *(acceleration_first + i);
        *(acceleration_first + i) =
          as<thrust::tuple<double, double>>(
            laplacian_constant * viscosities[type_i] * nl.reduce(
              i
              , pos_i
              , as<thrust::tuple<double, double>>(acc_i)
              , PARTICLE_LAMBDA_DEVICE(int i, int j, thrust::tuple<double, double>& value)
              {
                int type_j = *(type_first + j);
                if (categories[type_j] == static_cast<int>(category_t::ghost))
                  return;
                auto pos_j = *(position_first + j);
                auto vel_j = *(velocity_first + j);
                double r = particle::geometry::distance(pos_i, pos_j);
                value = as<thrust::tuple<double, double>>(
                  value + (vel_j - vel_i) * kernel(r));
              }));
      });

    double dt = ip.time_step;
    thrust::transform(
      col.begin<tag::velocity>()
      , col.end<tag::velocity>()
      , col.begin<tag::acceleration>()
      , col.begin<tag::pred_velocity>()
      , PARTICLE_LAMBDA(
        const thrust::tuple<double, double>& vel
        , const thrust::tuple<double, double>& acc)
      {
        return particle::geometry::as<thrust::tuple<double, double>>(vel + acc * dt);
      });

    thrust::transform(
      col.begin<tag::position>()
      , col.end<tag::position>()
      , col.begin<tag::pred_velocity>()
      , col.begin<tag::pred_position>()
      , PARTICLE_LAMBDA(
        const thrust::tuple<double, double>& pos
        ,const thrust::tuple<double, double>& vel)
      {
        return particle::geometry::as<thrust::tuple<double, double>>(pos + vel * dt);
      });

    cl.build(
      col.begin<tag::pred_position>()
      , col.end<tag::pred_position>()
      , col.begin<boost::mpl::vector<
      tag::id, tag::type, tag::pred_x, tag::pred_y, tag::pred_vx, tag::pred_vy>>());

    thrust::fill(col.begin<tag::pnd>(), col.end<tag::pnd>(), 0.0f);
    auto pnd_first = col.begin<tag::pnd>();
    auto pred_position_first = col.begin<tag::pred_position>();
    cl.pair_interact(
      col.begin<tag::pred_position>()
      , col.end<tag::pred_position>()
      , PARTICLE_LAMBDA_DEVICE(int i, cell_list::neighbour_list nl)
      {
        using particle::geometry::as;
        auto pos_i = *(pred_position_first + i);
        *(pnd_first + i) = nl.reduce(
          i
          , pos_i
          , 0.0
          , PARTICLE_LAMBDA_DEVICE(int i, int j, double& value)
          {
            auto pos_j = *(pred_position_first + j);
            double r = particle::geometry::distance(pos_i, pos_j);
            value = value + kernel(r);
          });
      });
    
    double c0 = ip.speed_of_sound;
    thrust::transform(
      col.begin<tag::pnd>()
      , col.end<tag::pnd>()
      , col.begin<tag::type>()
      , col.begin<tag::pressure>()
      , PARTICLE_LAMBDA_DEVICE(double pnd, int t)
      {
        if (categories[t] == static_cast<int>(category_t::ghost))
          return 0.0;
        return max(0.0, c0 * c0 * densities[t] * (pnd / pnd0 - 1)); 
      });

    double fst = ip.free_surface_threshold;
    double fsp = ip.free_surface_pressure;
    thrust::replace_if(
      col.begin<tag::pressure>()
      , col.end<tag::pressure>()
      , col.begin<tag::pnd>()
      , PARTICLE_LAMBDA(double pnd)
      {
        if (pnd < pnd0 * fst)
          return true;
        return false;
      }
      , fsp);

    thrust::fill(
      col.begin<tag::pressure_grad>()
      , col.end<tag::pressure_grad>()
      , thrust::make_tuple(0.0, 0.0));
    auto pressure_first = col.begin<tag::pressure>();
    auto pressure_grad_first = col.begin<tag::pressure_grad>();
    cl.pair_interact(
      col.begin<tag::pred_position>()
      , col.end<tag::pred_position>()
      , PARTICLE_LAMBDA_DEVICE(int i, cell_list::neighbour_list nl)
      {
        using particle::geometry::as;
        int type_i = *(type_first + i);
        if (categories[type_i] != static_cast<int>(category_t::fluid))
          return;
        auto pressure_i = *(pressure_first + i);
        auto pos_i = *(pred_position_first + i);
        *(pressure_grad_first + i) =
          as<thrust::tuple<double, double>>(
            gradient_constant / densities[type_i] * nl.reduce(
              i
              , pos_i
              , thrust::make_tuple(0.0, 0.0)
              , PARTICLE_LAMBDA_DEVICE(int i, int j, thrust::tuple<double, double>& value)
              {
                int type_j = *(type_first + j);
                if (categories[type_j] == static_cast<int>(category_t::ghost))
                  return;
                auto pressure_j = *(pressure_first + j);
                auto pos_j = *(pred_position_first + j);
                auto d = pos_j - pos_i;
                double rsq = particle::geometry::length_square(d);
                double r = sqrt(rsq);
                value = particle::geometry::as<thrust::tuple<double, double>>(
                  value + d * (pressure_j + pressure_i) / rsq * kernel(r));
              }));
      });

    thrust::transform(
      col.begin<tag::pred_velocity>()
      , col.end<tag::pred_velocity>()
      , col.begin<tag::pressure_grad>()
      , col.begin<tag::velocity>()
      , PARTICLE_LAMBDA(
        const thrust::tuple<double, double>& vel
        , const thrust::tuple<double, double>& grad)
      {
        return particle::geometry::as<thrust::tuple<double, double>>(vel - grad * dt);
      });

    thrust::transform(
      col.begin<tag::pred_position>()
      , col.end<tag::pred_position>()
      , col.begin<tag::pressure_grad>()
      , col.begin<tag::position>()
      , PARTICLE_LAMBDA(
        const thrust::tuple<double, double>& pos
        , const thrust::tuple<double, double>& grad)
      {
        return particle::geometry::as<thrust::tuple<double, double>>(pos - grad * dt * dt);
      });

    if ((step + 1) % ip.steps_per_output == 0)
    {
      thrust::copy(col.begin<output_tags>(), col.end<output_tags>(), h_col.begin<output_tags>());
      std::stringstream ss;
      ss << ip.output_filename << "_" << step + 1 << ".vtu";
      particle::io::vtk::save(
        ss.str()
        , h_col.begin<tag::position>()
        , h_col.end<tag::position>()
        , boost::fusion::make_vector("id", "type", "pnd", "pressure", "velocity")
        , boost::fusion::make_vector(
          h_col.begin<tag::id>()
          , h_col.begin<tag::type>()
          , h_col.begin<tag::pnd>()
          , h_col.begin<tag::pressure>()
          , h_col.begin<tag::velocity>()));
    }
  }

  return 0;
}
