#include "hip/hip_runtime.h"
// Boost test headers
#pragma push
#pragma diag_suppress = partial_override
#define BOOST_TEST_MODULE geometry
#include <boost/test/included/unit_test.hpp>
#pragma pop

// Particle headers
#include <particle/geometry/adapted/cuda/int2.hpp>
#include <particle/geometry/elem.hpp>

BOOST_AUTO_TEST_CASE(dim_test)
{
  using particle::geometry::traits::dim;

  constexpr int d = dim<int2>::value;
  BOOST_CHECK_EQUAL(d, 2);
}

BOOST_AUTO_TEST_CASE(elem_test)
{
  using particle::geometry::elem;

  using vec_type = int2;
  
  vec_type array = {1, 2};

  BOOST_CHECK_EQUAL(elem<0>(array), 1);
  BOOST_CHECK_EQUAL(elem<1>(array), 2);
}
