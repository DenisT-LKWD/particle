#include "hip/hip_runtime.h"
// Boost test headers
#pragma push
#pragma diag_suppress = partial_override
#define BOOST_TEST_MODULE geometry
#include <boost/test/included/unit_test.hpp>
#pragma pop

// Particle headers
#include <particle/geometry/adapted/cuda/float3.hpp>
#include <particle/geometry/elem.hpp>

BOOST_AUTO_TEST_CASE(dim_test)
{
  using particle::geometry::traits::dim;

  constexpr float d = dim<float3>::value;
  BOOST_CHECK_EQUAL(d, 3);
}

BOOST_AUTO_TEST_CASE(elem_test)
{
  using particle::geometry::elem;

  using vec_type = float3;
  
  vec_type array = {1.0f, 2.0f, 3.0f};

  const float e = 1e-12f;
  BOOST_CHECK_CLOSE(elem<0>(array), 1.0f, e);
  BOOST_CHECK_CLOSE(elem<1>(array), 2.0f, e);
  BOOST_CHECK_CLOSE(elem<2>(array), 3.0f, e);
}
